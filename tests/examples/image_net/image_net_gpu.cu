#include "hip/hip_runtime.h"
#include <iostream>

#include "cuda/include/hip/hip_runtime_api.h"
#include "examples/image_net/image_net_gpu.h"

namespace qcraft::nets {
namespace {

__global__ void ImageNetGpuKernel() {
  std::printf("Hello from ImageNetGPU Kernel\n");
}

}  // namespace

void ImageNetGPU::Inference() {
  ImageNetGpuKernel<<<1, 1>>>();
  std::printf("ImageNetGPU inference\n");
  hipDeviceSynchronize();
}

}  // namespace qcraft::nets
